/*
 * DeviceCache.cpp
 *
 *  Created on: Apr 5, 2013
 *      Author: jbarbosa
 */

#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include <config/common.h>
#include <gamalib/utils/x86_utils.h>

#include <boost/icl/interval.hpp>
#include <boost/icl/interval_set.hpp>
#include <boost/icl/split_interval_set.hpp>
#include <boost/icl/closed_interval.hpp>
#include <boost/icl/right_open_interval.hpp>
#include <boost/thread/mutex.hpp>
#include <boost/thread/thread.hpp>

#include "DeviceCache.h"

#include <cstdio>
#include <cstdlib>

using namespace std;
using namespace boost::icl;

DeviceCache::DeviceCache()
{
	//get the amount of free memory on the graphics card
	size_t free;
	size_t total;
	checkCudaErrors(hipMemGetInfo(&free,&total));

	if (free < CACHE_SIZE) {
		fprintf(stderr,"Out of GPU memory\n");
		exit(0);
	}

	checkCudaErrors(hipMalloc((void**)&cache,CACHE_SIZE));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(_cache),&cache,sizeof(void*)));

	dcache = cache;
}

DeviceCache::~DeviceCache()
{
	// TODO Auto-generated destructor stub
}

bool DeviceCache::cachePtr(void* ptr, size_t size, hipStream_t* st) {
	boost::mutex::scoped_lock lock(_lock);
	boost::icl::right_open_interval<_ulong> r((_ulong)ptr,(_ulong)ptr+size);

	if(boost::icl::contains(mem,r)) {
		return true;
	}

	if(!(boost::icl::intersects(mem,r))) {
		copyInterval(r,st);
		return true;
	}

	boost::icl::interval_set<_ulong, std::less, right_open_interval<_ulong> > list = findIntervals(r);

	for(interval_set<_ulong, std::less, right_open_interval<_ulong> >::iterator memloc = list.begin(); memloc != list.end(); ++memloc) {
			copyInterval(*memloc,st);
	}

	return true;
}

void DeviceCache::cacheReset() {
	mem.clear();
}

bool DeviceCache::copyInterval(right_open_interval<_ulong> r, hipStream_t* st) {


	void *ptrFrom = (void*) r.lower();
	void *ptrTo = (void*) ((unsigned long long) cache + (r.lower() & offset_filter));

//	printf("Copying\n");
	_ulong size = r.upper() - r.lower();

	checkCudaErrors(hipMemcpyAsync(ptrTo, // dst
					ptrFrom,// src
					size,// size
					hipMemcpyHostToDevice, *st// stream
					));
//	boost::mutex::scoped_lock lock(_lock);
	mem.add(r);
	return true;
}

void DeviceCache::printCache() {
	cout << mem << endl;
}

boost::icl::interval_set<_ulong, std::less, right_open_interval<_ulong> > DeviceCache::findIntervals(boost::icl::right_open_interval<_ulong> r) {
//	boost::mutex::scoped_lock lock(_lock);

	boost::icl::interval_set<_ulong, std::less, right_open_interval<_ulong> > toCache;
	right_open_interval<_ulong> elem = r;

	for(interval_set<_ulong, std::less, right_open_interval<_ulong> >::iterator memloc = mem.begin(); memloc != mem.end() && !is_empty(elem); ++memloc) {
		right_open_interval<_ulong> elem2 = right_subtract(elem, *memloc);
		elem = left_subtract(elem, *memloc);
		if(!is_empty(elem2) && elem2 != elem) {
			toCache.add(elem2);
		}
	}

	if(!is_empty(elem)) {
		toCache.add(elem);
	}
	return toCache;
}

#include "hip/hip_runtime.h"
/*
 * DeviceCuda.cu
 *
 *  Created on: Apr 11, 2012
 *      Author: jbarbosa
 */

#include <hip/hip_runtime.h>

#include <config/common.h>

#include <gamalib/memlib/LowLevelMemAllocator.h>

#include "DeviceCuda.h"

#include <config/vtable.h>
#include <config/vtable.cuh>
#include <gamalib/utils/cuda_utils.cuh>
#include <gamalib/GenericKernels/KernelCuda.cuh>
#include <gamalib/utils/x86_utils.h>

#define PostWork(W) outbox->enqueue(W);

typedef unsigned char byte;

__global__
//__launch_bounds__(256, 5)
void genericKernel(
		Workqueue<work, INBOX_QUEUE_SIZE, GPU_CUDA>* INBOX) {

	unsigned long index = blockIdx.x * blockDim.x + threadIdx.x;
	work* w_item;

	if ((w_item = INBOX->data[index]) != NULL) {
		(w_item->*WORK_GPU_TABLE[w_item->getWorkTypeID()])();
	}

}
__global__
//__launch_bounds__(256, 5)
void genericWideKernel(work* w_item) {
	(w_item->*WORK_GPU_TABLE[w_item->getWorkTypeID()])();
}

__inline__ void CudaTest(char *msg) {
	hipError_t e;

	if (hipSuccess != (e = hipGetLastError())) {
		fprintf(stderr, "%s: %d\n", msg, e);
		fprintf(stderr, "%s\n", hipGetErrorString(e));
		exit(-1);
	}
}
void DeviceCuda::classWideKernel(work* witem, unsigned st) {
	hipSetDevice(cudaDeviceID);

#if defined GAMA_CACHE
	std::vector<pointerInfo>* Lw = (witem->*TOCACHEW_CPU_TABLE[witem->getWorkTypeID()])();
	std::vector<pointerInfo>* Lr = (witem->*TOCACHER_CPU_TABLE[witem->getWorkTypeID()])();

	for(int i=0; i< Lr->size(); i++) {
		cache->cachePtr((*Lr)[i].ptr,(*Lr)[i].lenght,&streams[st]);
	}


#endif

	if (!IS_LIB_CALL(witem->WORK_TYPE_ID)) {
		genericWideKernel<<< cdp.multiProcessorCount * 5, 256, 0, streams[st]>>>(witem);
		CudaTest("Launching parallel wide kernel");

	} else {

		(witem->*WORK_GPU_LIB_TABLE[witem->getWorkTypeID()])();
	}

	CudaTest("Launching parallel wide kernel");

#if defined GAMA_CACHE

	delete Lw;
	delete Lr;
#endif
}

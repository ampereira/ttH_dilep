#include "hip/hip_runtime.h"
#include "neut.h"


using std::vector;
using namespace std;

namespace Dilep {
	namespace GPU {

		unsigned _tid = 0;
	
		__device__
		double calcMass (double x, double y, double z, double e) {
			double mm, mass;

			mm = e*e - (x*x + y*y + z*z);

			if (mm < 0.0)
				mass = -sqrt(-mm);
			else
				mass = sqrt(mm);

			return mass;
		}

		__device__
		void calcMass (double array[]) {
			double mm = array[3]*array[3] - (array[0]*array[0] + array[1]*array[1] + array[2]*array[2]);

			if (mm < 0.0)
				array[4] = -sqrt(-mm);
			else
				array[4] = sqrt(mm);
		}

		// TLorentzs/Flags
		// [0] -> x
		// [1] -> y
		// [2] -> z
		// [3] -> E/isb
		// [4] -> M

		__device__
		void applyVariance (double _in_mpx[], double _in_mpy[], double _z_lepWFlags[], double _c_lepWFlags[],
			double _z_bjWFlags[], double _c_bjWFlags[], double _z_lep[], double _c_lep[], double _z_bj[], double _c_bj[],
			double _z_bl[], double _c_bl[], double _MissPx, double _MissPy, unsigned size) {

			//unsigned tid = _tid;
			unsigned tid = threadIdx.x + blockIdx.x * blockDim.x;

			// Using pointers for better code readbility - does it affect the performance in the kernel?
			double *in_mpx		= &CUDA_THREAD_STRIDE2(_in_mpx, 0);
			double *in_mpy		= &CUDA_THREAD_STRIDE2(_in_mpy, 0);
			double *z_lepWFlags = &CUDA_THREAD_STRIDE5(_z_lepWFlags, 0);
			double *c_lepWFlags = &CUDA_THREAD_STRIDE5(_c_lepWFlags, 0);
			double *z_bjWFlags	= &CUDA_THREAD_STRIDE5(_z_bjWFlags, 0);
			double *c_bjWFlags  = &CUDA_THREAD_STRIDE5(_c_bjWFlags, 0);
			double *z_lep 		= &CUDA_THREAD_STRIDE5(_z_lep, 0);
			double *c_lep 		= &CUDA_THREAD_STRIDE5(_c_lep, 0);
			double *z_bj 		= &CUDA_THREAD_STRIDE5(_z_bj, 0);
			double *c_bj 		= &CUDA_THREAD_STRIDE5(_c_bj, 0);
			double *z_bl 		= &CUDA_THREAD_STRIDE5(_z_bl, 0);
			double *c_bl 		= &CUDA_THREAD_STRIDE5(_c_bl, 0);

			
			// new four-vectors	
			/*double n_Px, n_Py, n_Pz, n_Pt, n_E;	
			double delPx, delPy;

			double prng_nums[3];
			gaus_kernel( 0., RESOLUTION, &prng_nums[0], state );
			gaus_kernel( 0., RESOLUTION, &prng_nums[1], state );
			gaus_kernel( 0., RESOLUTION, &prng_nums[2], state );

			// Vary!

			// _______________________________
			// _______z_lep___________________
			// _______________________________
			if (  abs(  z_lepWFlags[3]  )  ==  11  ){ //___electrons____
				n_Px = z_lepWFlags[0] * ( 1. + prng_nums[0] );
				n_Py = z_lepWFlags[1] * ( 1. + prng_nums[1] );
				n_Pz = z_lepWFlags[2] * ( 1. + prng_nums[2] );
			} else if (  abs(z_lepWFlags[3]) == 13 ){ //_____muons______
				n_Px = z_lepWFlags[0] * ( 1. + prng_nums[0] );
				n_Py = z_lepWFlags[1] * ( 1. + prng_nums[1] );
				n_Pz = z_lepWFlags[2] * ( 1. + prng_nums[2] );
			}
			// Recalculate z_lep
			n_E = sqrt ( n_Px*n_Px + n_Py*n_Py + n_Pz*n_Pz + z_lepWFlags[4]*z_lepWFlags[4] );
			z_lep[0] = n_Px;	// Change Px 				
			z_lep[1] = n_Py; 	// Change Py 	
			z_lep[2] = n_Pz; 	// Change Pz 
			z_lep[3] =  n_E; 	// Change E 
			// Propagate to MissPx and MissPy
			delPx = z_lepWFlags[0] - n_Px; 
			delPy = z_lepWFlags[1] - n_Py;			
			in_mpx[0] = _MissPx + delPx; in_mpx[1] = _MissPx + delPx; // initialize miss(Px,Py) neutrino 1
			in_mpy[0] = _MissPy + delPy; in_mpy[1] = _MissPy + delPy; // initialize miss(Px,Py) neutrino 2

			// _______________________________
			// _______c_lep___________________
			// _______________________________
			gaus_kernel( 0., RESOLUTION, &prng_nums[0], state );
			gaus_kernel( 0., RESOLUTION, &prng_nums[1], state );
			gaus_kernel( 0., RESOLUTION, &prng_nums[2], state );

			if (  abs(  c_lepWFlags[3]  )  ==  11  ){ //___electrons____
				n_Px = c_lepWFlags[0] * ( 1. + prng_nums[0] );
				n_Py = c_lepWFlags[1] * ( 1. + prng_nums[1] );
				n_Pz = c_lepWFlags[2] * ( 1. + prng_nums[2] );
			} else if (  abs(c_lepWFlags[3]) == 13 ){ //_____muons______
				n_Px = c_lepWFlags[0] * ( 1. + prng_nums[0] );
				n_Py = c_lepWFlags[1] * ( 1. + prng_nums[1] );
				n_Pz = c_lepWFlags[2] * ( 1. + prng_nums[2] );
			}
			// Recalculate c_lep
			n_E = sqrt ( n_Px*n_Px + n_Py*n_Py + n_Pz*n_Pz + c_lepWFlags[4]*c_lepWFlags[4] );
			c_lep[0] = n_Px;	// Change Px 				
			c_lep[1] = n_Py; 	// Change Py 	
			c_lep[2] = n_Pz; 	// Change Pz 
			c_lep[3] = n_E ; 	// Change E 
			// Propagate to MissPx and MissPy
			delPx = c_lepWFlags[0] - n_Px; 
			delPy = c_lepWFlags[1] - n_Py;			
			in_mpx[0] += delPx; in_mpx[1] += delPx; // correct miss(Px,Py) neutrino 1
			in_mpy[0] += delPy; in_mpy[1] += delPy; // correct miss(Px,Py) neutrino 2

			// _______________________________
			// _______z_bj____________________
			// _______________________________
			gaus_kernel( 0., RESOLUTION, &prng_nums[0], state );
			gaus_kernel( 0., RESOLUTION, &prng_nums[1], state );
			gaus_kernel( 0., RESOLUTION, &prng_nums[2], state );

			n_Px = z_bjWFlags[0] * ( 1. + prng_nums[0] );
			n_Py = z_bjWFlags[1] * ( 1. + prng_nums[1] );
			n_Pz = z_bjWFlags[2] * ( 1. + prng_nums[2] );
			// Recalculate z_bj
			n_E = sqrt ( n_Px*n_Px + n_Py*n_Py + n_Pz*n_Pz + z_bjWFlags[4]*z_bjWFlags[4] );
			z_bj[0] = n_Px;	// Change Px 				
			z_bj[1] = n_Py; 	// Change Py 	
			z_bj[2] = n_Pz; 	// Change Pz 
			z_bj[3] = n_E ; 	// Change E 
			// Propagate to MissPx and MissPy
			delPx = z_bjWFlags[0] - n_Px; 
			delPy = z_bjWFlags[1] - n_Py;			
			in_mpx[0] += delPx; in_mpx[1] += delPx; // correct miss(Px,Py) neutrino 1
			in_mpy[0] += delPy; in_mpy[1] += delPy; // correct miss(Px,Py) neutrino 2


			// _______________________________
			// _______c_bj____________________
			// _______________________________
			gaus_kernel( 0., RESOLUTION, &prng_nums[0], state );
			gaus_kernel( 0., RESOLUTION, &prng_nums[1], state );
			gaus_kernel( 0., RESOLUTION, &prng_nums[2], state );

			n_Px = c_bjWFlags[0] * ( 1. + prng_nums[0] );
			n_Py = c_bjWFlags[1] * ( 1. + prng_nums[1] );
			n_Pz = c_bjWFlags[2] * ( 1. + prng_nums[2] );
		//	n_Pt = c_bjWFlags.Pt() * ( 1. + _t_rnd_.Gaus( 0., St_j ) );
		//	n_E  = c_bjWFlags.E()  * ( 1. + _t_rnd_.Gaus( 0., Se_j ) );
			// Recalculate c_bj
			n_E = sqrt ( n_Px*n_Px + n_Py*n_Py + n_Pz*n_Pz + c_bjWFlags[4]*c_bjWFlags[4] );
			c_bj[0] = n_Px;	// Change Px 				
			c_bj[1] = n_Py; 	// Change Py 	
			c_bj[2] = n_Pz; 	// Change Pz 
			c_bj[3] = n_E ; 	// Change E 
			// Propagate to MissPx and MissPy
			delPx = c_bjWFlags[0] - n_Px; 
			delPy = c_bjWFlags[1] - n_Py;			
			in_mpx[0] += delPx; in_mpx[1] += delPx; // correct miss(Px,Py) neutrino 1
			in_mpy[0] += delPy; in_mpy[1] += delPy; // correct miss(Px,Py) neutrino 2
*/
			// ---------------------------------------
			// Define TLorentzVectors for (b,l) system
			// ---------------------------------------
			//z_bl = z_bj + z_lep;
			//c_bl = c_bj + c_lep;
			

			z_bl[tid * 5 + 0] = z_bj[tid * 5 + 0] + z_lep[tid * 5 + 0];
			z_bl[tid * 5 + 1] = z_bj[tid * 5 + tid * 5 + 1] + z_lep[tid * 5 + tid * 5 + 1];
			z_bl[tid * 5 + 2] = z_bj[tid * 5 + tid * 5 + 2] + z_lep[tid * 5 + tid * 5 + 2];
			z_bl[tid * 5 + 3] = z_bj[tid * 5 + tid * 5 + 3] + z_lep[tid * 5 + tid * 5 + 3];

			c_bl[tid * 5 + 0] = c_bj[tid * 5 + 0] + c_lep[tid * 5 + 0];
			c_bl[tid * 5 + 1] = c_bj[tid * 5 + 1] + c_lep[tid * 5 + 1];
			c_bl[tid * 5 + 2] = c_bj[tid * 5 + 2] + c_lep[tid * 5 + 2];
			c_bl[tid * 5 + 3] = c_bj[tid * 5 + 3] + c_lep[tid * 5 + 3];

			// Re-calculate the masses
			calcMass(&z_bl[tid * 5]);
			calcMass(&c_bl[tid * 5]);
		}

		__global__
		void dilep_kernel (double _in_mpx[], double _in_mpy[], double _z_lepWFlags[], double _c_lepWFlags[],
			double _z_bjWFlags[], double _c_bjWFlags[], double _z_lep[], double _c_lep[], double _z_bj[], double _c_bj[],
			double *_MissPx, double *_MissPy, double _t_mass[], double _w_mass[], double nc[], int a[], unsigned *size) {

			// CPU version
			//double _z_bl[5 * size], _c_bl[5 * size];

			//for (_tid = 0; _tid < size; ++_tid)
			//	applyVariance(_in_mpx, _in_mpy, _z_lepWFlags, _c_lepWFlags, _z_bjWFlags, _c_bjWFlags,
			//		_z_lep, _c_lep, _z_bj, _c_bj, _z_bl, _c_bl, _MissPx, _MissPy);
			
			//for (unsigned tid = 0; tid < size; ++tid)
			//	calc_dilep(_t_mass, _w_mass, _in_mpx, _in_mpy, 
			//				_z_lep, _c_lep, _z_bl, _c_bl, nc, a, tid);

			// GPU version

			//applyVariance(_in_mpx, _in_mpy, _z_lepWFlags, _c_lepWFlags, _z_bjWFlags, _c_bjWFlags,
			//		_z_lep, _c_lep, _z_bj, _c_bj, _z_bl, _c_bl, *_MissPx, *_MissPy, *size);

			calc_dilep(_t_mass, _w_mass, _in_mpx, _in_mpy, 
							_z_lep, _c_lep, _z_bj, _c_bj, nc, a);
		}


		__host__
		void dilep (vector<DilepInput> &di) {

			unsigned size = di.size();
			double in_mpx[2 * size], in_mpy[2 * size], t_mass[2 * size], w_mass[2 * size];
			double a[5 * size], b[5 * size], c[5 * size], d[5 * size]; 
			double aFlags[5 * size], bFlags[5 * size], cFlags[5 * size], dFlags[5 * size];
			double nc[16*size];
			int hasSolution = 0, count[size];
		
			unsigned *dev_size;
			double *dev_t_mass, *dev_w_mass, *dev_in_mpx, *dev_in_mpy;
			double *dev_lep_a, *dev_lep_b, *dev_bj_a, *dev_bj_b;
			double *dev_lep_aFlags, *dev_lep_bFlags, *dev_bj_aFlags, *dev_bj_bFlags;
			double *dev_nc, *dev_MissPx, *dev_MissPy;
			int *dev_count;

			double _misspx = di[0].getMissPx();
			double _misspy = di[0].getMissPy();

			// time measurement
			#ifdef MEASURE_DILEP
			long long int time = startTimer();
			#endif

			for (unsigned i = 0; i < size; ++i) {

				in_mpx[i * 2]		= di[i].getInMpx(0);
				in_mpx[(i * 2) + 1] = di[i].getInMpx(1);
				in_mpy[i * 2]		= di[i].getInMpy(0);
				in_mpy[(i * 2) + 1] = di[i].getInMpy(1);
				t_mass[i * 2]		= di[i].getTmass(0);
				t_mass[(i * 2) + 1] = di[i].getTmass(1);
				w_mass[i * 2]		= di[i].getWmass(0);
				w_mass[(i * 2) + 1] = di[i].getWmass(1);
				
				// z_lep
				a[i * 5]	   = di[i].getZlep().Px();
				a[(i * 5) + 1] = di[i].getZlep().Py();
				a[(i * 5) + 2] = di[i].getZlep().Pz();
				a[(i * 5) + 3] = di[i].getZlep().E();
				a[(i * 5) + 4] = di[i].getZlep().M();

				// z_lepWFlags
				aFlags[i * 5]	    = di[i].getZlepW().Px();
				aFlags[(i * 5) + 1] = di[i].getZlepW().Py();
				aFlags[(i * 5) + 2] = di[i].getZlepW().Pz();
				aFlags[(i * 5) + 3] = di[i].getZlepW().isb;
				aFlags[(i * 5) + 4] = di[i].getZlepW().M();

				// c_lep
				b[i * 5]	   = di[i].getClep().Px();
				b[(i * 5) + 1] = di[i].getClep().Py();
				b[(i * 5) + 2] = di[i].getClep().Pz();
				b[(i * 5) + 3] = di[i].getClep().E();
				b[(i * 5) + 4] = di[i].getClep().M();

				// c_lepWFlags
				bFlags[i * 5]	    = di[i].getClepW().Px();
				bFlags[(i * 5) + 1] = di[i].getClepW().Py();
				bFlags[(i * 5) + 2] = di[i].getClepW().Pz();
				bFlags[(i * 5) + 3] = di[i].getClepW().isb;
				bFlags[(i * 5) + 4] = di[i].getClepW().M();

				// z_bj
				c[i * 5]	   = di[i].getZbj().Px();
				c[(i * 5) + 1] = di[i].getZbj().Py();
				c[(i * 5) + 2] = di[i].getZbj().Pz();
				c[(i * 5) + 3] = di[i].getZbj().E();
				c[(i * 5) + 4] = di[i].getZbj().M();

				// z_bjWFlags
				cFlags[i * 5]	    = di[i].getZbjW().Px();
				cFlags[(i * 5) + 1] = di[i].getZbjW().Py();
				cFlags[(i * 5) + 2] = di[i].getZbjW().Pz();
				cFlags[(i * 5) + 3] = di[i].getZbjW().isb;
				cFlags[(i * 5) + 4] = di[i].getZbjW().M();

				// c_bj
				d[i * 5]	   = di[i].getCbj().Px();
				d[(i * 5) + 1] = di[i].getCbj().Py();
				d[(i * 5) + 2] = di[i].getCbj().Pz();
				d[(i * 5) + 3] = di[i].getCbj().E();
				d[(i * 5) + 4] = di[i].getCbj().M();

				// c_bjWFlags
				dFlags[i * 5]	    = di[i].getCbjW().Px();
				dFlags[(i * 5) + 1] = di[i].getCbjW().Py();
				dFlags[(i * 5) + 2] = di[i].getCbjW().Pz();
				dFlags[(i * 5) + 3] = di[i].getCbjW().isb;
				dFlags[(i * 5) + 4] = di[i].getCbjW().M();
			}

			unsigned tamG, tamB;
			if (size > 192) {
				tamG = (size) / 192;
				tamB = 192;
			} else {
				tamG = 1;
				tamB = size;
			}

			// GPU memory allocation of the inputs and outputs of the dilep kernel
			hipMalloc(&dev_t_mass, size*2*sizeof(double));
			hipMalloc(&dev_w_mass, size*2*sizeof(double));
			hipMalloc(&dev_in_mpx, size*2*sizeof(double));
			hipMalloc(&dev_in_mpy, size*2*sizeof(double));

			hipMalloc(&dev_lep_a, sizeof(a));
			hipMalloc(&dev_lep_b, sizeof(b));
			hipMalloc(&dev_bj_a, sizeof(c));
			hipMalloc(&dev_bj_b, sizeof(d));


			hipMalloc(&dev_lep_aFlags, sizeof(aFlags));
			hipMalloc(&dev_lep_bFlags, sizeof(bFlags));
			hipMalloc(&dev_bj_aFlags, sizeof(cFlags));
			hipMalloc(&dev_bj_bFlags, sizeof(dFlags));

			hipMalloc(&dev_MissPx, sizeof(double));
			hipMalloc(&dev_MissPy, sizeof(double));

			hipMalloc(&dev_size, sizeof(unsigned));

			// allocation of the results
			hipMalloc(&dev_nc, size*16*sizeof(double));
			hipMalloc(&dev_count, size*sizeof(int));

			
			// transfer the inputs to GPU memory
			hipMemcpy(dev_t_mass, t_mass, sizeof(t_mass), hipMemcpyHostToDevice);
			hipMemcpy(dev_w_mass, w_mass, sizeof(w_mass), hipMemcpyHostToDevice);
			hipMemcpy(dev_in_mpx, in_mpx, sizeof(in_mpx), hipMemcpyHostToDevice);
			hipMemcpy(dev_in_mpy, in_mpy, sizeof(in_mpy), hipMemcpyHostToDevice);

			hipMemcpy(dev_lep_a, a, sizeof(a), hipMemcpyHostToDevice);
			hipMemcpy(dev_lep_b, b, sizeof(b), hipMemcpyHostToDevice);
			hipMemcpy(dev_bj_a, c, sizeof(c), hipMemcpyHostToDevice);
			hipMemcpy(dev_bj_b, d, sizeof(d), hipMemcpyHostToDevice);

			hipMemcpy(dev_lep_aFlags, aFlags, sizeof(aFlags), hipMemcpyHostToDevice);
			hipMemcpy(dev_lep_bFlags, bFlags, sizeof(bFlags), hipMemcpyHostToDevice);
			hipMemcpy(dev_bj_aFlags, cFlags, sizeof(cFlags), hipMemcpyHostToDevice);
			hipMemcpy(dev_bj_bFlags, dFlags, sizeof(dFlags), hipMemcpyHostToDevice);

			hipMemcpy(dev_MissPx, &_misspx, sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(dev_MissPy, &_misspy, sizeof(double), hipMemcpyHostToDevice);
			
			hipMemcpy(dev_size, &size, sizeof(unsigned), hipMemcpyHostToDevice);


			dim3 grid_size1D (tamG);
			dim3 block_size1D (tamB);

			dilep_kernel <<< grid_size1D, block_size1D >>> (dev_in_mpx, dev_in_mpy, dev_lep_aFlags, dev_lep_bFlags, dev_bj_aFlags, dev_bj_bFlags,
					dev_lep_a, dev_lep_b, dev_bj_a, dev_bj_b, dev_MissPx, dev_MissPy, dev_t_mass, dev_w_mass, dev_nc, dev_count, dev_size);
			
			//calc_dilep <<< grid_size1D, block_size1D >>> (dev_t_mass, dev_w_mass, dev_in_mpx, dev_in_mpy, dev_lep_a, dev_lep_b,
			//		dev_bj_a, dev_bj_b, dev_nc, dev_count);

			
			// memory transfer of the results from the GPU

			hipMemcpy(count, dev_count, size*sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(nc, dev_nc, 16*size*sizeof(double), hipMemcpyDeviceToHost);


			// reconstruction of the normal output of dilep
			// o num de combs*vars e o num de threads

			for (unsigned comb = 0; comb < size; ++comb) {
				vector<myvector> result;

				for (int sol = 0 ; sol < count[comb] && sol<4 ; sol++) {
					myvector *mv = new myvector( 
						TO1D(nc,comb,sol,0),
						TO1D(nc,comb,sol,1),
						TO1D(nc,comb,sol,2),
						TO1D(nc,comb,sol,3) );
					result.push_back(*mv);
				}
				if(result.size())
					++hasSolution;

				di[comb].setHasSol(hasSolution);
				di[comb].setResult(&result);
			}

			hipFree(dev_t_mass);
			hipFree(dev_w_mass);
			hipFree(dev_in_mpx);
			hipFree(dev_in_mpy);
			hipFree(dev_lep_a);
			hipFree(dev_lep_b);
			hipFree(dev_bj_a);
			hipFree(dev_bj_b);
			hipFree(dev_lep_aFlags);
			hipFree(dev_lep_bFlags);
			hipFree(dev_bj_aFlags);
			hipFree(dev_bj_bFlags);
			hipFree(dev_MissPx);
			hipFree(dev_MissPy);
			hipFree(dev_size);
			hipFree(dev_nc);
			hipFree(dev_count);

			// time measurement
			#ifdef MEASURE_DILEP
			stopTimer(time);
			#endif
			
		}

		__device__
		void calc_dilep(double t_mass[], double w_mass[], 
				double in_mpx[], double in_mpy[], double _lep_a[], 
				double _lep_b[], double _bl_a[], double _bl_b[], 
				double nc[], int a[])
		{

			unsigned tid = threadIdx.x + blockIdx.x * blockDim.x;
			//unsigned tid = 1;
			double G_1, G_3;
			double WMass_a, WMass_b, tMass_a, tMass_b, lep_a[5], lep_b[5], bl_a[5], bl_b[5], bj_a[5], bj_b[5];
			double in_mpz[2] = {0.0, 0.0};


			WMass_a = STRIDE2(w_mass, 0);
			tMass_a = STRIDE2(t_mass, 0);
			WMass_b = STRIDE2(w_mass, 1);
			tMass_b = STRIDE2(t_mass, 1);

			for (unsigned i = 0; i < 5; ++i) {
				lep_a[i] = STRIDE5(_lep_a, i);
				lep_b[i] = STRIDE5(_lep_b, i);

				bj_a[i] = STRIDE5(_bl_a, i);
				bj_b[i] = STRIDE5(_bl_b, i);
			}
			
			bl_a[0] = bj_a[0] + lep_a[0];
			bl_a[1] = bj_a[1] + lep_a[1];
			bl_a[2] = bj_a[2] + lep_a[2];
			bl_a[3] = bj_a[3] + lep_a[3];

			bl_b[0] = bj_b[0] + lep_b[0];
			bl_b[1] = bj_b[1] + lep_b[1];
			bl_b[2] = bj_b[2] + lep_b[2];
			bl_b[3] = bj_b[3] + lep_b[3];
			

			G_1 = (WMass_a - lep_a[4]) * (WMass_a + lep_a[4]);
			G_3 = (WMass_b - lep_b[4]) * (WMass_b + lep_b[4]);

			double G_5,G_6,G_7,G_8,G_9,G_10,G_11,G_12;
			G_5 = ( bl_a[0]/bl_a[3] - lep_a[0]/lep_a[3] );
			G_6 = ( bl_a[1]/bl_a[3] - lep_a[1]/lep_a[3] );
			G_7 = ( bl_a[2]/bl_a[3] - lep_a[2]/lep_a[3] );
			G_8 = ( G_1/lep_a[3] - ((tMass_a - bl_a[4]) * (tMass_a + bl_a[4]))/bl_a[3] )/2.;

			G_9 =	( bl_b[0]/bl_b[3] - lep_b[0]/lep_b[3] );
			G_10 =	( bl_b[1]/bl_b[3] - lep_b[1]/lep_b[3] );
			G_11 =	( bl_b[2]/bl_b[3] - lep_b[2]/lep_b[3] );
			G_12 =	( G_3/lep_b[3] - ((tMass_b - bl_b[4]) * (tMass_b + bl_b[4]))/bl_b[3] )/2.;

			///////////////////////////////////////////////////////////////////
			//// 	G_5 *x1 + G_6*y1 + G_7*z1 = G8;  		(6)
			////  	G_9 *x2 + G_10*y2 + G_11*z2 = G12; 		(7)
			////  	2*El_1*sqrt() - 2*(ax1+by1+cz1) = G_1;  	(8)
			////  	2*El_2*sqrt() - 2*(Ax2+By2+Cz2) = G_3;		(9)
			////  	x1+x2 = S;					(10)
			////  	y1+y2 = T;					(11)
			////  	bring z1 and z2 (from 6/7) to 7 and 8
			///////////////////////////////////////////////////////////////////

			//// 1st top decay product /////
			
			double in_a[5],out_a[6];
			in_a[0] = G_8/G_7;
			in_a[1] = -1.0*G_5/G_7;
			in_a[2] = -1.0*G_6/G_7;
			in_a[3] = lep_a[3];
			in_a[4] = G_1;
			toz(in_a, lep_a, out_a);

			double in_c[5],out_c[6];
			in_c[0] = G_12/G_11;
			in_c[1] = -1*G_9/G_11;
			in_c[2] = -1*G_10/G_11;
			in_c[3] = lep_b[3];
			in_c[4] = G_3;
			toz(in_c, lep_b, out_c);
			/////////////////////////////////////////////////////
			//////change x2 y2 equation to x1 and y1 by using
			////// 		x1+x2 = S = in_mpx[0]
			////// 		y1+y2 = T = mpy
			/////////////////////////////////////////////////////
			double out_e[6];
			out_e[0] = out_c[0];
			out_e[1] = out_c[1];
			out_e[2] = -1*( out_c[0]*in_mpx[0] + out_c[2] + out_c[4]*in_mpy[0]);
			out_e[3] = -1*( out_c[1]*in_mpy[0] + out_c[3] + out_c[4]*in_mpx[0]);
			out_e[4] = out_c[4]; 
			out_e[5] =( out_c[0]*in_mpx[0]*in_mpx[0] + out_c[1]*in_mpy[0]*in_mpy[0] + 2*out_c[2]*in_mpx[0] + 2*out_c[3]*in_mpy[0] + out_c[5] + 2*out_c[4]*in_mpx[0]*in_mpy[0]);

			///////////////////////////////////////////////////
			///  solve 
			/// {ax2+by2+2dx+2ey+2fxy+g=0		(12)
			/// {Ax2+By2+2Dx+2Ey+2Fxy+G=0		(13)
			/// out_a[6]: 0   1    2    3    4     5
			/// out_a[6]: a   b    d    e    f     g
			/// out_e[6]: A   B    D    E    F     G
			/// if a!=0, everything is OK.
			///
			/// if a==0, then we can get x2 = f(x,y) from (13)
			/// (12) --> [x2 - f(x,y)] + by2 + ... = 0
			///////////////////////////////////////////////////
			
			double fx_1, fx_2, fx_3, fx_4, fx_5;
			double k_1, k_2, k_3, k_4, k_5;

			// bad organization of the code; diminished register spilling
			if ( out_a[0]!=0  ){
				fx_1 = 2.*(out_e[0]*out_a[2] - out_a[0]*out_e[2]);
				fx_2 = 2.*(out_e[0]*out_a[4] - out_a[0]*out_e[4]);
				fx_3 = out_a[0]*out_e[1] - out_e[0]*out_a[1];
				fx_4 = 2.*(out_e[3]*out_a[0] - out_e[0]*out_a[3]);
				fx_5 = out_a[0]*out_e[5] - out_a[5]*out_e[0];

				k_1 = ( out_a[4]*out_a[4] - out_a[0]*out_a[1] )/out_a[0]/out_a[0];
				k_2 = ( 2.*out_a[2]*out_a[4] - 2.*out_a[0]*out_a[3] )/out_a[0]/out_a[0];
				k_3 = ( out_a[2]*out_a[2]-out_a[0]*out_a[5] )/out_a[0]/out_a[0];
				k_4 = -out_a[2]/out_a[0];
				k_5 = -out_a[4]/out_a[0];
			} else {
				if (out_a[0]==0 && out_e[0]!=0 ) {
					fx_1 = 2.*(out_e[0]*(out_a[2] + out_e[2]/out_e[0]) - out_e[2]);
					fx_2 = 2.*(out_e[0]*(out_a[4] + out_e[4]/out_e[0]) - out_e[4]);
					fx_3 = out_e[1] - out_e[0]*(out_a[1] + out_e[1]/out_e[0]);
					fx_4 = 2.*(out_e[3] - out_e[0]*(out_a[3] + out_e[3]/out_e[0]));
					fx_5 = out_e[5] - (out_a[5] + out_e[5]/out_e[0])*out_e[0];

					k_1 = ( (out_a[4] + out_e[4]/out_e[0])*(out_a[4] + out_e[4]/out_e[0]) - (out_a[1] + out_e[1]/out_e[0]) );
					k_2 = ( 2.*(out_a[2] + out_e[2]/out_e[0])*(out_a[4] + out_e[4]/out_e[0]) - 2.*(out_a[3] + out_e[3]/out_e[0]) );
					k_3 = ( (out_a[2] + out_e[2]/out_e[0])*(out_a[2] + out_e[2]/out_e[0])-(out_a[5] + out_e[5]/out_e[0]) );
					k_4 = -(out_a[2] + out_e[2]/out_e[0]);
					k_5 = -(out_a[4] + out_e[4]/out_e[0]);
				}
			}

			if ( out_a[0]==0 && out_e[0]==0){
				return;
			}


			/////
			///// the part above is 
			///// x = (fx3*y**2 + fx4*y + fx5)/(fx1 + fx2*y)
			///// used to get x value once y is known
			/////
			//// if fx1 + fx2*y == 0, then x is 
			//// x = +/-sqrt(k1*y**2 + k2*y + k3) + (k4 + k5*y)
			////


			double g_1 = 4.*out_e[0]*out_e[0]*k_5*k_5 + 4.*out_e[4]*out_e[4] + 8.*out_e[0]*out_e[4]*k_5;
			double m_1 = g_1*k_1;
			double g_2 = 8.*out_e[0]*out_e[0]*k_4*k_5 + 8.*out_e[0]*out_e[2]*k_5 + 8.*out_e[0]*out_e[4]*k_4 + 8.*out_e[2]*out_e[4];
			double g_3 = 4.*out_e[0]*out_e[0]*k_4*k_4 + 4.*out_e[2]*out_e[2] + 8.*out_e[0]*out_e[2]*k_4;
			double g_4 = out_e[0]*k_1 + out_e[0]*k_5*k_5;
			double g_5 = out_e[0]*k_2 + 2.*out_e[0]*k_4*k_5 + 2.*out_e[2]*k_5;
			double g_6 = out_e[0]*k_3 + out_e[0]*k_4*k_4 + 2.*out_e[2]*k_4 + out_e[5];

			double m_2 = g_1*k_2 + g_2*k_1;
			double m_3 = g_1*k_3 + g_2*k_2 + g_3*k_1;
			double m_4 = g_2*k_3 + g_3*k_2;
			double m_5 = g_3*k_3;

			double m_6  = out_e[1]*out_e[1] + 4.*out_e[4]*out_e[4]*k_5*k_5 + 4.*out_e[1]*out_e[4]*k_5;
			double m_7  = 4.*out_e[1]*out_e[3] + 8.*out_e[4]*out_e[4]*k_4*k_5 + 4.*out_e[1]*out_e[4]*k_4 + 8.*out_e[3]*out_e[4]*k_5;
			double m_8  = 4.*out_e[3]*out_e[3] + 4.*out_e[4]*out_e[4]*k_4*k_4 + 8.*out_e[3]*out_e[4]*k_4;
			double m_80 = pow(g_4,2);
			double m_81 = 2*g_4*g_5;
			double m_9  = pow(g_5,2) + 2.*g_4*g_6;
			double m_10 = 2.*g_5*g_6;
			double m_11 = g_6*g_6;

			double m_12 = 	2.*out_e[0]*out_e[1]*k_1 + 2.*out_e[0]*out_e[1]*k_5*k_5 + 4.*out_e[0]*out_e[4]*k_1*k_5 + 4.*out_e[0]*out_e[4]*pow(k_5,3);
			double m_13 = 	2.*out_e[0]*out_e[1]*k_2 + 4.*out_e[0]*out_e[1]*k_4*k_5 + 4.*out_e[1]*out_e[2]*k_5 + 
				4.*out_e[0]*(out_e[3]*k_1 + out_e[3]*k_5*k_5 + out_e[4]*k_1*k_4 + out_e[4]*k_2*k_5) + 
				12.*out_e[0]*out_e[4]*k_4*k_5*k_5 + 8.*out_e[2]*out_e[4]*k_5*k_5;
			double m_14 = 	2.*out_e[0]*out_e[1]*k_3 + 2.*out_e[0]*out_e[1]*k_4*k_4 + 4.*out_e[2]*out_e[1]*k_4 + 2.*out_e[1]*out_e[5] + 4.*out_e[0]*out_e[3]*k_2 + 
				8.*out_e[0]*out_e[3]*k_4*k_5 + 8.*out_e[3]*out_e[2]*k_5 + 4.*out_e[0]*out_e[4]*k_2*k_4 + 4.*out_e[0]*out_e[4]*k_3*k_5 + 
				12.*out_e[0]*out_e[4]*k_4*k_4*k_5 + 16.*out_e[2]*out_e[4]*k_4*k_5 + 4.*out_e[4]*out_e[5]*k_5;
			double m_15 = 	4.*out_e[0]*out_e[3]*(k_3 + k_4*k_4) + 8.*out_e[3]*out_e[2]*k_4 + 4.*out_e[3]*out_e[5] + 4.*out_e[0]*out_e[4]*(k_3*k_4 + pow(k_4,3)) + 
				8.*out_e[2]*out_e[4]*k_4*k_4 + 4.*out_e[4]*out_e[5]*k_4;

			double  re[5];
			re[0] = m_1 - m_6 - m_12 - m_80;
			re[1] = m_2 - m_7 - m_13 - m_81;
			re[2] = m_3 - m_8 - m_9 - m_14;
			re[3] = m_4 - m_10 - m_15;
			re[4] = m_5 - m_11;  



			double output[8];
			my_qu(re, output);

			int ncand(0);

			double rec_x1, rec_y1, rec_z1, rec_e1, rec_x2, rec_y2, rec_z2, rec_e2;

			for (int j=0; j<8; j+=2){
				double delta = k_1*output[j]*output[j] + k_2*output[j] + k_3;
				if ( output[j+1]==0 && delta >=0) {
					if ( (fx_1 + fx_2*output[j])!=0 ) {
						rec_x1 = (fx_3*pow(output[j],2) + fx_4*output[j] + fx_5)/(fx_1 + fx_2*output[j]);
					} else {
						rec_x1 = sqrt(delta)+k_4+k_5*output[j];
					}  

					rec_y1 = output[j];
					rec_z1 = G_8/G_7 - G_5*rec_x1/G_7 - G_6*rec_y1/G_7;
					rec_e1 = sqrt(rec_x1*rec_x1 + rec_y1*rec_y1 + rec_z1*rec_z1);
					rec_x2 = in_mpx[0] - rec_x1;
					rec_y2 = in_mpy[0] - rec_y1;
					rec_z2 = G_12/G_11 - G_9*rec_x2/G_11 - G_10*rec_y2/G_11;
					rec_e2 = sqrt(rec_x2*rec_x2 + rec_y2*rec_y2 + rec_z2*rec_z2);
					
					// self-consistence check and control of the solutions

					double m_w11 = calcMass(rec_x1+lep_a[0], rec_y1+lep_a[1], rec_z1+lep_a[2], rec_e1+lep_a[3]);
					double m_w12 = calcMass(rec_x2+lep_b[0], rec_y2+lep_b[1], rec_z2+lep_b[2], rec_e2+lep_b[3]);
					double m_t11 = calcMass(rec_x1+ bl_a[0], rec_y1+ bl_a[1], rec_z1+ bl_a[2], rec_e1+ bl_a[3]);
					double m_t12 = calcMass(rec_x2+ bl_b[0], rec_y2+ bl_b[1], rec_z2+ bl_b[2], rec_e2+ bl_b[3]);

					// m_delta_mass is 1000.0
					bool m_good_eq1 = ( fabs(in_mpx[0] -(rec_x1+rec_x2)) <= 0.01 ) * true + 
									  ( fabs(in_mpx[0] -(rec_x1+rec_x2)) > 0.01 ) * false;
					bool m_good_eq2 = ( fabs(in_mpy[0] -(rec_y1+rec_y2)) <= 0.01 ) * true +
									  ( fabs(in_mpy[0] -(rec_y1+rec_y2)) > 0.01 ) * false;
					bool m_good_eq3 = ( fabs(m_w11 - w_mass[0]) <= 1000.0 ) * true + 
									  ( fabs(m_w11 - w_mass[0]) > 1000.0 ) * false;
					bool m_good_eq4 = ( fabs(m_w12 - w_mass[1]) <= 1000.0 ) * true +
									  ( fabs(m_w12 - w_mass[1]) > 1000.0 ) * false;
					bool m_good_eq5 = ( fabs(m_t11 - t_mass[0]) <= 1000.0 ) * true +
									  ( fabs(m_t11 - t_mass[0]) > 1000.0 ) * false;
					bool m_good_eq6 = ( fabs(m_t12 - t_mass[1]) <= 1000.0 ) * true +
									  ( fabs(m_t12 - t_mass[1]) <= 1000.0 ) * false;

					bool cond = m_good_eq1 && m_good_eq2 && m_good_eq3 && m_good_eq4 && m_good_eq5 && m_good_eq6;
					
					// aqui podem nao chegar as threads todas
					//__syncthreads();
					nc[tid * 16 + 2*j] = cond * rec_x1;
					nc[tid * 16 + 2*j + 1] = cond * rec_y1;
					nc[tid * 16 + 2*j + 2] = cond * rec_z1;
					nc[tid * 16 + 2*j + 3] = cond * rec_z2;
					ncand += cond * 1;
				}
			}

			// indicates the number of solutions that this thread found
			a[tid] = ncand;
		}

		



		//////////////////////////////////////
		__device__
		void toz(double k[], double l[], double g[]){
			//// checked !!
			///////////////////////////////////////////////////////////////////////////
			///// bring z=A+Bx+Cy to 2*D*sqrt(x**2+y**2+z**2)-2(ax+by+dz) = E
			///// simplify it to g1*x^2 + g2*y^2 + 2*g3*x + 2*g4*y + 2*g5*xy + g6 = 0
			///////////////////////////////////////////////////////////////////////////
			double A = k[0];
			double B = k[1];
			double C = k[2];
			double D = k[3];
			double E = k[4];
			double a = l[0];
			double b = l[1];
			double d = l[2];
			g[0] = 4*pow(D,2)*( 1 + pow(B,2)) - 4*pow(a,2) -4*pow(d,2)*pow(B,2) - 8*a*d*B;
			if ( g[0]!=0 ) { 
				g[1] = ( 4*pow(D,2)*( 1 + pow(C,2)) - 4*pow(b,2) -4*pow(d,2)*pow(C,2) - 8*b*d*C )/g[0] ;
				g[2] = ( (4*pow(D,2)-4*d*d)*A*B - 4*a*d*A - 2*E*a - 2*E*d*B )/g[0];
				g[3] = ( (4*pow(D,2)-4*d*d)*A*C - 4*b*d*A - 2*E*b - 2*E*d*C )/g[0];
				g[4] = ( (4*pow(D,2)-4*d*d)*B*C - 4*a*b   - 4*a*d*C - 4*b*d*B )/g[0];
				g[5] = ( (4*pow(D,2)-4*d*d)*A*A - E*E - 4*E*d*A )/g[0];
				g[0] = 1.0; 
			} else {
				g[1] = ( 4*pow(D,2)*( 1 + pow(C,2)) - 4*pow(b,2) -4*pow(d,2)*pow(C,2) - 8*b*d*C ) ;
				g[2] = ( (4*pow(D,2)-4*d*d)*A*B - 4*a*d*A - 2*E*a - 2*E*d*B );
				g[3] = ( (4*pow(D,2)-4*d*d)*A*C - 4*b*d*A - 2*E*b - 2*E*d*C );
				g[4] = ( (4*pow(D,2)-4*d*d)*B*C - 4*a*b   - 4*a*d*C - 4*b*d*B );
				g[5] = ( (4*pow(D,2)-4*d*d)*A*A - E*E - 4*E*d*A );
				g[0] = 0.;  
			}	 
			return;
		}


		///////////////////////////////////////////
		__device__
		void my_qu( double my_in[], double my_val[])
		{

			///////////////////////////////////////////
			/////  find the solution /////////////////
			/////  ax^4+bx^3+cx^2+dx+e=0
			//////////////////////////////////////////
			double a=my_in[0];
			double b=my_in[1];
			double c=my_in[2];
			double d=my_in[3];
			double e=my_in[4];

			double real[3]={0,0,0};
			double img[3]={0,0,0};
			double x1_r = 0; double x1_i = 0;
			double x2_r = 0; double x2_i = 0;
			double x3_r = 0; double x3_i = 0;
			double x4_r = 0; double x4_i = 0;


			/////////////////////////////////////////////
			///// in case of a==0, simplify to cubic
			///// bx^3+cx^2+dx+e=0
			/////////////////////////////////////////////

			if ( a ==0 && b!=0 ){
				double input[4]={b,c,d,e};
				cubic(input,real,img);
				x1_r = real[0];	x1_i = img[0];
				x2_r = real[1];	x2_i = img[1];
				x3_r = real[2];	x3_i = img[2];
				my_val[0] = x1_r;
				my_val[1] = x1_i;
				my_val[2] = x2_r;
				my_val[3] = x2_i;
				my_val[4] = x3_r;
				my_val[5] = x3_i;
				my_val[6] = x4_r;
				my_val[7] = x4_i;
				
				return;
			}


			/////////////////////////////////////////////
			///// in case of a==0 && b==0, 
			///// simplify to quadratic
			///// cx*2 + d*x + e =0
			/////////////////////////////////////////////

			if ( a ==0 && b==0 && c!=0){
				double alpha = pow(d,2)-4*c*e;
				if (alpha>=0) {
					x1_r = (-1*d + sqrt(alpha))/2/c;	x1_i = 0;
					x2_r = (-1*d - sqrt(alpha))/2/c;	x2_i = 0;
				} else {
					x1_r = (-1*d )/2/c;	x1_i = sqrt(-alpha)/2/c;
					x2_r = (-1*d )/2/c;	x2_i = -1*sqrt(-alpha)/2/c;  	
				}
				my_val[0] = x1_r;
				my_val[1] = x1_i;
				my_val[2] = x2_r;
				my_val[3] = x2_i;
				my_val[4] = x3_r;
				my_val[5] = x3_i;
				my_val[6] = x4_r;
				my_val[7] = x4_i;
				
				return;
			}


			/////////////////////////////////////////////
			///// in case of a==0 && b==0 && c==0, 
			///// simplify to linear equation dx + e =0
			/////////////////////////////////////////////

			if (a ==0 && b==0 && c==0 && d!=0){
				x1_r = -e/d;	x1_i = 0;
				my_val[0] = x1_r;
				my_val[1] = x1_i;
				my_val[2] = x2_r;
				my_val[3] = x2_i;
				my_val[4] = x3_r;
				my_val[5] = x3_i;
				my_val[6] = x4_r;
				my_val[7] = x4_i;
				
				return;	
			}


			//////////////////////////////////////////////
			////  (1)the common expression
			///////////////////////////////////////////////
			double bb=b/a; double cc=c/a;
			double dd=d/a; double ee=e/a;

			//////////////////////////////////////////////
			////  (2) the equation changes to 
			////	x^4 + bb*x^3 + cc*x^2 + dd*x + ee=0
			////
			////  (3) substitude x=y-aa/4, then we get
			////    y^4 + f*y^2 + g*y + h =0; where
			///////////////////////////////////////////

			double _f = cc - 3*pow(bb,2)/8;
			double _g = dd + (pow(bb,3)/8) - (bb*cc/2);
			double _h = ee - (3*pow(bb,4)/256) + (pow(bb,2)*cc/16) - (bb*dd/4);

			////////////////////////////////////////////////////////////////////////////
			///  (4) the normal situation is f, g and h are non-zero; then		////
			//// 	the related cubic equation is					////
			//// 	z^3 + (f/2) z^2 + ( (f^2-4h)/16 )*z -g^2/64 =0;			////
			////	 it has three "squared" roots, for example, p,q and l, 		////
			//// 	then p^2, q^2 and l^2 are the root of equation above		////
			//// 	set r=-g/8(pq), then the four roots of the original quartic are	////
			//// 		x = p + q + r -bb/4;					////
			//// 		x = p - q - r -bb/4;					////
			//// 		x = -p + q - r -bb/4;					////
			//// 		x = -p - q + r -bb/4;					////
			////////////////////////////////////////////////////////////////////////////

			double c_1 = 1.;
			double c_2 = _f/2;
			double c_3 = (pow(_f,2)-4*_h)/16.;
			double c_4 = -1*pow(_g,2)/64.;

			double input[4]={c_1,c_2,c_3,c_4};
			cubic(input,real,img);

			////////////////////////////////////////////////
			////// (5) sqrt root of the cubic equation solutions
			////////////////////////////////////////////////

			double out_r[3]={0,0,0};
			double out_i[3]={0,0,0};
			bool _img[3]={false,false,false};
			int img_index =0;
			double my[2];

			for (int ii=0; ii<3; ii++){
				Csqrt(real[ii],img[ii],my);
				out_r[ii] = my[0];
				out_i[ii] = my[1];
				if ( my[1]!=0 ) {
					_img[ii]=true;
					img_index++;
				}
			}    


			///////////////////////////////////////////////
			////(x + yi)(u + vi) = (xu - yv) + (xv + yu)i
			//// calculating r = -g/(8pq)
			////////////////////////////////////////////////
			double r_r;	double r_i;
			double p_r;	double p_i;
			double q_r;	double q_i;

			if (_img[0]==_img[1] && out_r[0]*out_r[1]>0 ){
				r_r	= out_r[0]*out_r[1] - out_i[0]*out_i[1];
				r_i	= out_r[0]*out_i[1] + out_r[1]*out_i[0];
				p_r	= out_r[0]; p_i = out_i[0];
				q_r	= out_r[1]; q_i = out_i[1];
			} else {	
				for (int kk=0; kk<2; kk++) {
					for (int k=kk+1; k<3; k++) {
						if (_img[kk]==_img[k]){ 
							r_r	= out_r[kk]*out_r[k] - out_i[kk]*out_i[k];
							r_i	= out_r[kk]*out_i[k] + out_r[k]*out_i[kk];
							p_r	= out_r[kk]; p_i = out_i[kk];
							q_r	= out_r[k]; q_i = out_i[k];
						}
					}
				}
			}


			if( r_r !=0 )	{ 
				r_r 	= -1.*_g/8/r_r;
			} else {
				r_r 	= 0;
			}
			if( r_i !=0 )	{ 
				r_r 	= -1.*_g/8/r_i;
			} else {
				r_i 	= 0;
			}	

			x1_r = p_r + q_r + r_r -b/a/4;
			x1_i = p_i + q_i + r_i;
			x2_r = p_r - q_r - r_r -b/a/4;
			x2_i = p_i - q_i - r_i;
			x3_r = -1*p_r + q_r - r_r -b/a/4;
			x3_i = -1*p_i + q_i - r_i;
			x4_r = -1*p_r - q_r + r_r -b/a/4;
			x4_i = -1*p_i - q_i + r_i;

			my_val[0] = x1_r;
			my_val[1] = x1_i;
			my_val[2] = x2_r;
			my_val[3] = x2_i;
			my_val[4] = x3_r;
			my_val[5] = x3_i;
			my_val[6] = x4_r;
			my_val[7] = x4_i;
		}
		////////////////////end of main
		///////////////////////////////////////////////////////////////
		////+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
		__device__
		void Csqrt(double _ar, double _ai, double _my[])
		{
			///// complex sqrt
			double x,y,r,w;
			if  (  (_ar == 0.0) && (_ai == 0.0) ) {
				_my[0]=0.0;
				_my[1]=0.0;
				return;
			} else {	
				x=fabs(_ar);
				y=fabs(_ai);
				if (x >= y) {
					r=y/x;
					w=sqrt(x)*sqrt(0.5*(1.0+sqrt(1.0+r*r)));
				} else {
					r=x/y;
					w=sqrt(y)*sqrt(0.5*(r+sqrt(1.0+r*r)));
				}
				if (_ar>= 0.0) {
					_my[0]=w;
					_my[1]=_ai/(2.0*w);
				} else {
					_my[1]=(_ai >= 0) ? w : -w;
					_my[0]=_ai/(2.0*_my[1]);
				}
				return;
			}
		}

		//////////////////////////////////////////////////////////////////
		/// cubic /// a[0]x^3+a[1]x^2+a[2]x+a[3]=0
		//////////////////////////////////////////////////////////////////
		__device__
		void cubic(double a[], double rr[], double ri[])
		{
			int i;
			double a0, a1, a2, a3;
			double g, h, y1, sh, theta, xy1, xy2, xy3;
			double y2, z1, z2, z3, z4;
			//// initialize the results
			for (i = 0; i < 3; i ++)
			{
				rr[i] = 0.0;
				ri[i] = 0.0;
			}

			a0 = a[0];
			a1 = a[1]/3.0;
			a2 = a[2]/3.0;
			a3 = a[3];

			g = (a0 * a0) * a3 - 3.0 * a0 * a1 * a2 + 2.0 * pow(a1, 3);
			h = a0 * a2 - a1 * a1;
			y1 = g * g + 4.0 * pow(h, 3);

			if (y1 < 0.0){
				sh = sqrt(-h);
				theta = acos(g / (2.0 * h * sh)) / 3.0;
				xy1 = 2.0 * sh * cos(theta);
				xy2 = 2.0 * sh * cos(theta + (2.0 * TPI / 3.0));
				xy3 = 2.0 * sh * cos(theta + (4.0 * TPI / 3.0));
				rr[0] = (xy1 - a1) / a0;
				rr[1] = (xy2 - a1) / a0;
				rr[2] = (xy3 - a1) / a0;
				return;
			} else {
				y2 = sqrt(y1);
				z1 = (g + y2) / 2.0;
				z2 = (g - y2) / 2.0;
				if (z1 < 0.0){
					z3 = pow(-z1, 1.0/3.0);
					z3 = -z3;
				} else  z3 = pow(z1, 1.0/3.0);
				if (z2 < 0.0){
					z4 = pow(-z2, 1.0/3.0);
					z4 = - z4;
				}
				else  z4 = pow(z2, 1.0/3.0);

				rr[0] = -(a1 + z3 + z4) / a0;
				rr[1] = (-2.0 * a1 + z3 + z4) / (2.0 * a0);
				ri[1] = sqrt(3.0) * (z4 - z3) / (2.0 * a0);
				rr[2] = rr[1];
				ri[2] = -ri[1];

				return;

			}
		}
	}
}

